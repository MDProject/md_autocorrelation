#include "hip/hip_runtime.h"
#include "/home/xdengae/xdrfile-1.1.4/include/xdrfile_xtc.h"
#include "/home/xdengae/xdrfile-1.1.4/include/xdrfile_trr.h"
#include "/home/xdengae/xdrfile-1.1.4/include/xdrfile.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "correlation_lib.h"

#define threadsPerBlock 8
#define blocksPerGrid 8

using namespace std;
#define PI 3.14159265359

void handleError(hipError_t cu) {
	if (cu != hipSuccess) {
		printf("%s\n", hipGetErrorString(cu));
		system("pause");
		exit(0);
	}
}

__global__ void calCorrelationXKernel(float kx,float kz,int numOfFrame,int numOfAtom, float* vx,float* vz,float* coor_x,float* coor_z,int deltatIdx,float* corXup,float* corXdown,float* corZup,float* corZdown,float H,float alpha){
    // corX : store the partial summation of correlation function for kz at different frame      deltatIdx:    time interval's index dt=deltatIdx*delta_t
    // using shared memory to calculate the sum(corXFrame)
    // blockIdx --> frameIdx    threadIdx --> particle idx
    // corXPerFrame1 --> accumulate the summation of vx(z)[t0] * vec_x(z)[t0] of all atoms; corXPerFrame2 --> accumulate the summation of vx(z)[t0+dt] * vec_x(z)[t0+dt]
	__shared__ float corXPerFrame1[threadsPerBlock]; // cacheIdx12
	__shared__ float corXPerFrame2[threadsPerBlock]; // cacheIdx12
    __shared__ float corZPerFrame1[threadsPerBlock]; // cacheIdx12
	__shared__ float corZPerFrame2[threadsPerBlock];
	float corXFrameTmp_up = 0.;
	float corXFrameTmp_down = 0.;
    float corZFrameTmp_up = 0.;
	float corZFrameTmp_down = 0.;
	int frameIdx = blockIdx.x;
	while (frameIdx<numOfFrame - deltatIdx) {
        int cacheIdx12 = threadIdx.x;
        float corXPerFrame1Tmp = 0.;
	    float corXPerFrame2Tmp = 0.;
        float corZPerFrame1Tmp = 0.;
        float corZPerFrame2Tmp = 0.;
		while (cacheIdx12<numOfAtom) {
			int atomIdx_t0 = frameIdx*numOfAtom + cacheIdx12;
			int atomIdx_deltat = (frameIdx + deltatIdx)*numOfAtom + cacheIdx12;
			corXPerFrame1Tmp += vx[atomIdx_t0] * velocityX(kx, kz, coor_x[atomIdx_t0], coor_z[atomIdx_t0],H,alpha);
			corXPerFrame2Tmp += vx[atomIdx_deltat] * velocityX(kx, kz, coor_x[atomIdx_deltat], coor_z[atomIdx_deltat],H,alpha);
            corZPerFrame1Tmp += vz[atomIdx_t0] * velocityZ(kx, kz, coor_x[atomIdx_t0], coor_z[atomIdx_t0],H,alpha);
			corZPerFrame2Tmp += vz[atomIdx_deltat] * velocityZ(kx, kz, coor_x[atomIdx_deltat], coor_z[atomIdx_deltat],H,alpha);
			cacheIdx12 += blockDim.x;
		}
		corXPerFrame1[threadIdx.x] = corXPerFrame1Tmp;
		corXPerFrame2[threadIdx.x] = corXPerFrame2Tmp;
        corZPerFrame1[threadIdx.x] = corZPerFrame1Tmp;
		corZPerFrame2[threadIdx.x] = corZPerFrame2Tmp;
		__syncthreads();
		if (threadIdx.x == 0) {
			float sum_vx_t0 = 0.;
			float sum_vx_delta = 0.;
            float sum_vz_t0 = 0.;
            float sum_vz_delta = 0.;
			for (int i = 0; i < threadsPerBlock; i++) {
				sum_vx_t0 += corXPerFrame1[i];
				sum_vx_delta += corXPerFrame2[i];
                sum_vz_t0 += corZPerFrame1[i];
                sum_vz_delta += corZPerFrame2[i];
			}
			corXFrameTmp_up += sum_vx_t0*sum_vx_delta;
            corXFrameTmp_down += sum_vx_t0*sum_vx_t0;
            corZFrameTmp_up += sum_vz_t0*sum_vz_delta;
            corZFrameTmp_down += sum_vz_t0*sum_vz_t0;
		}
		__syncthreads();
		frameIdx += gridDim.x;
	}
	if (threadIdx.x == 0) {
        corXup[blockIdx.x] = corXFrameTmp_up;
        corXdown[blockIdx.x]=corXFrameTmp_down;
        corZup[blockIdx.x]=corZFrameTmp_up;
        corZdown[blockIdx.x]=corZFrameTmp_down;
        //printf("%f\t%f\n",corXFrameTmp_up,corXFrameTmp_down);
	}
}

int main(int argc,char** argv){ // first argument: startFrame fraction   2nd argument: endFrame
    char path[]="/home/xdengae/LJ_Fluid/DATA/traj.trr"; // *.trr file
    // prepare host memory to store the particle's info
    int numOfAtoms=0;
    int numOfFrames=0;
    getNumOfFrameAtom(path,&numOfAtoms,&numOfFrames);
    int startFrame=numOfFrames*atof(argv[1]);
    int endFrame=numOfFrames*atof(argv[2]);
    numOfFrames=endFrame-startFrame+1;
    float* vx=(float*)calloc(numOfAtoms*numOfFrames,sizeof(float));
    float* vy=(float*)calloc(numOfAtoms*numOfFrames,sizeof(float));
    float* vz=(float*)calloc(numOfAtoms*numOfFrames,sizeof(float));
    float* coor_x=(float*)calloc(numOfAtoms*numOfFrames,sizeof(float));
    float* coor_z=(float*)calloc(numOfAtoms*numOfFrames,sizeof(float));
    float* corXFrameUp=(float*)calloc(blocksPerGrid,sizeof(float));
    float* corXFrameDown=(float*)calloc(blocksPerGrid,sizeof(float));
    float* corZFrameUp=(float*)calloc(blocksPerGrid,sizeof(float));
    float* corZFrameDown=(float*)calloc(blocksPerGrid,sizeof(float));
    float H,L;
    extract_trr_file(path,vx,vy,vz,coor_x,coor_z,startFrame,endFrame,&H,&L);
    unsigned int memsize_cpu=(5*numOfAtoms*numOfFrames*sizeof(float)+4*blocksPerGrid*sizeof(float))/1000000.;
    unsigned int memsize_gpu=(4*numOfAtoms*numOfFrames*sizeof(float)+4*blocksPerGrid*sizeof(float))/1000000.;
    printf("Atom's info takes up around %d MB memory space on host RAM and %d MB memory on GPU\n",memsize_cpu,memsize_gpu);
    // copy particle info from host to device
    float* dev_vx,* dev_vz,* dev_coor_x,* dev_coor_z,* dev_corXFrameUp,* dev_corXFrameDown,* dev_corZFrameUp,* dev_corZFrameDown;
    handleError(hipMalloc(&dev_vx,numOfAtoms*numOfFrames*sizeof(float)));
    handleError(hipMalloc(&dev_vz,numOfAtoms*numOfFrames*sizeof(float)));
    handleError(hipMalloc(&dev_coor_x,numOfAtoms*numOfFrames*sizeof(float)));
    handleError(hipMalloc(&dev_coor_z,numOfAtoms*numOfFrames*sizeof(float)));
    handleError(hipMalloc(&dev_corXFrameUp,blocksPerGrid*sizeof(float)));
    handleError(hipMalloc(&dev_corXFrameDown,blocksPerGrid*sizeof(float)));
    handleError(hipMalloc(&dev_corZFrameUp,blocksPerGrid*sizeof(float)));
    handleError(hipMalloc(&dev_corZFrameDown,blocksPerGrid*sizeof(float)));
    handleError(hipMemcpy(dev_coor_x,coor_x,numOfAtoms*numOfFrames*sizeof(float),hipMemcpyHostToDevice));
    handleError(hipMemcpy(dev_coor_z,coor_z,numOfAtoms*numOfFrames*sizeof(float),hipMemcpyHostToDevice));
    handleError(hipMemcpy(dev_vx,vx,numOfAtoms*numOfFrames*sizeof(float),hipMemcpyHostToDevice));
    handleError(hipMemcpy(dev_vz,vz,numOfAtoms*numOfFrames*sizeof(float),hipMemcpyHostToDevice));
    // launch GPU test kernel
    H-=3.6;
    dim3 grid(blocksPerGrid,1);
    dim3 block(threadsPerBlock,1);
    float kx_test=PI/L;
    float kz_test=0.01;
    int deltatIdx_test=4;
    calCorrelationXKernel<<<grid,block>>>(kx_test,kz_test,numOfFrames,numOfAtoms,dev_vx,dev_vz,dev_coor_x,dev_coor_z,deltatIdx_test,dev_corXFrameUp,dev_corXFrameDown,dev_corZFrameUp,dev_corZFrameDown,H,PI/2);
    handleError(hipMemcpy(corXFrameUp,dev_corXFrameUp,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
    handleError(hipMemcpy(corXFrameDown,dev_corXFrameDown,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
    handleError(hipMemcpy(corZFrameUp,dev_corZFrameUp,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
    handleError(hipMemcpy(corZFrameDown,dev_corZFrameDown,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
    float corX_up=0.;
    float corX_down=0.;
    float corZ_up=0.;
    float corZ_down=0.;
    for(int i=0;i<blocksPerGrid;i++){
        corX_up+=corXFrameUp[i];
        corX_down+=corXFrameDown[i];
        corZ_up+=corZFrameUp[i];
        corZ_down+=corZFrameDown[i];
    }
    float corX=corX_up/corX_down;
    float corZ=corZ_up/corZ_down;
    if(calCorrelationXKernel_TEST(kx_test,kz_test,numOfFrames,numOfAtoms,vx,vz,coor_x,coor_z,deltatIdx_test,corX,corZ,H,PI/2)){
        cout<<"GPU kernel test PASS"<<endl;
    }else{
        cout<<"GPU kernel test FAIL"<<endl;
        exit(0);
    }
    // temperature analysis
    float sum_kinetic=0.;
    float* temperature_table=(float*)malloc(numOfFrames*sizeof(float));
    for(int i=0;i<numOfFrames;i++){
        for(int j=0;j<numOfAtoms;j++){
            int idx=i*numOfAtoms+j;
            sum_kinetic+=vx[idx]*vx[idx]+vy[idx]*vy[idx]+vz[idx]*vz[idx];
        }
        float tmp=sum_kinetic/3./numOfAtoms/(i+1);
        temperature_table[i]=tmp;
    }
    sum_kinetic=sum_kinetic/3./numOfAtoms/numOfFrames;
    cout<<"frames average temperature: "<<sum_kinetic<<endl;
    write_temperature_per_frame(temperature_table,numOfFrames);
    free(temperature_table);
    // start to calculate the correaltion function versus time interval deltat
    cout<<"velocity X correaltion function versus time interval : "<<endl;
    for(int i=0;i<20;i++){
        calCorrelationXKernel<<<grid,block>>>(kx_test,kz_test,numOfFrames,numOfAtoms,dev_vx,dev_vz,dev_coor_x,dev_coor_z,i,dev_corXFrameUp,dev_corXFrameDown,dev_corZFrameUp,dev_corZFrameDown,H,PI/2);
        handleError(hipMemcpy(corXFrameUp,dev_corXFrameUp,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
        handleError(hipMemcpy(corXFrameDown,dev_corXFrameDown,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost));
        float corX_up=0.;
        float corX_down=0.;
        for(int i=0;i<blocksPerGrid;i++){
            corX_up+=corXFrameUp[i];
            corX_down+=corXFrameDown[i];
        }
        float corX_gpu=corX_up/corX_down;
        //float corX_cpu=calCorrelationXKernel_CPU(kx_test,kz_test,numOfFrames,numOfAtoms,vx,vz,coor_x,coor_z,i,H,PI/2);
        cout<<corX_gpu<<endl;
    }
}
